#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


#ifdef BL_USE_FLOAT
    typedef float Real;
#else
    typedef double Real;
#endif

typedef Real* pointertype;
typedef Real sizetype;


__global__ void gpu_jacobi(
pointertype x, 
pointertype b,
pointertype alpha, 
pointertype alphasing, 
#if (BL_SPACEDIM==3)
pointertype Uz,pointertype Lz,
#endif
pointertype Ux,pointertype Uy,  
pointertype Lx, pointertype Ly, 
int zstride, int ystride, int blocksInY, pointertype Ax,
pointertype rhs,pointertype soln,pointertype red,pointertype black,
pointertype mask,long int N,int NX,int NY,int NZ) {

unsigned int blockIdxz = blockIdx.y / blocksInY;
unsigned int blockIdxy = blockIdx.y % blocksInY;
unsigned int k = blockIdxz *blockDim.z + threadIdx.z;
unsigned int j = blockIdxy *blockDim.y + threadIdx.y;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
int idx = k*zstride + j*ystride + i;

 rhs[idx]=0.0;
 soln[idx]=0.0;
 red[idx]=0.0;
 black[idx]=0.0;
 Ax[idx]=0.0;
 __syncthreads();
 
 rhs[idx]=b[idx]-alphasing[idx]*x[idx]+
#if (BL_SPACEDIM==3)
  Uz[idx]*x[idx+zstride]+Lz[idx]*x[idx-zstride]+
#endif
  Ux[idx]*x[idx+1]+Lx[idx]*x[idx-1]+
  Uy[idx]*x[idx+ystride]+Ly[idx]*x[idx-ystride];

 __syncthreads();

 red[idx]=rhs[idx]/alpha[idx]; 
 __syncthreads();

 black[idx]=(rhs[idx]+
#if (BL_SPACEDIM==3)
  Uz[idx]*red[idx+zstride]+Lz[idx]*red[idx-zstride]+
#endif
  Ux[idx]*red[idx+1]+Lx[idx]*red[idx-1]+
  Uy[idx]*red[idx+ystride]+Ly[idx]*red[idx-ystride])/alpha[idx];
 __syncthreads();

 red[idx]=(rhs[idx]+
#if (BL_SPACEDIM==3)
  Uz[idx]*black[idx+zstride]+Lz[idx]*black[idx-zstride]+
#endif 
  Ux[idx]*black[idx+1]+Lx[idx]*black[idx-1]+
  Uy[idx]*black[idx+ystride]+Ly[idx]*black[idx-ystride])/alpha[idx];
 __syncthreads();

 soln[idx]=mask[idx]*red[idx]+(1.0-mask[idx])*black[idx];
 __syncthreads();

 Ax[idx]=x[idx]+soln[idx];
 __syncthreads();
}

extern "C" void kernel_gpu_jacobi(
 pointertype x, 
 pointertype b, 
 pointertype xx_d,
 pointertype bb_d,
 pointertype ax_d,
 pointertype rhs_d,
 pointertype soln_d,
 pointertype red_d,
 pointertype black_d,
 pointertype mask_d,
 pointertype alpha_d,
 pointertype alphasing_d,
#if (BL_SPACEDIM==3)
 pointertype Uz_d,pointertype Lz_d,
#endif
 pointertype Ux_d, pointertype Uy_d, 
 pointertype Lx_d, pointertype Ly_d, 
 long int Np, int NX, int NY, int NZ) {

    long int N=Np;
    int zstride=NX*NY;
    int ystride=NX;
    long int grid_bytes=sizeof(sizetype)*N;

      /* grid dimensions divisible by 8 */
    int blocksIny = NY/4;
    dim3 dimBlock(512,1,1);
    dim3 dimGrid(NX/8,NZ*NY/64,1);

    hipMemcpy(xx_d, x, grid_bytes, hipMemcpyHostToDevice);
    hipMemcpy(bb_d, b, grid_bytes, hipMemcpyHostToDevice);

    gpu_jacobi <<< dimGrid, dimBlock >>> (xx_d,bb_d,
      alpha_d,alphasing_d, 
#if (BL_SPACEDIM==3)
      Uz_d,Lz_d,
#endif
      Ux_d, Uy_d, 
      Lx_d, Ly_d, 
      zstride,ystride, blocksIny, ax_d,
      rhs_d,soln_d,red_d,black_d,mask_d,
      N,NX,NY,NZ);

    hipMemcpy(x, ax_d, grid_bytes, hipMemcpyDeviceToHost);
}


__global__ void gpu_apply(
pointertype x, 
pointertype alpha, 
#if (BL_SPACEDIM==3)
pointertype Uz,pointertype Lz,
#endif
pointertype Ux,pointertype Uy, 
pointertype Lx, pointertype Ly, 
int zstride, int ystride, int blocksInY, pointertype Ax,
long int N,int NX,int NY,int NZ)
{
unsigned int blockIdxz = blockIdx.y / blocksInY;
unsigned int blockIdxy = blockIdx.y % blocksInY;
unsigned int k = blockIdxz *blockDim.z + threadIdx.z;
unsigned int j = blockIdxy *blockDim.y + threadIdx.y;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
int idx = k*zstride + j*ystride + i;

 Ax[idx]=alpha[idx]*x[idx]-(
#if (BL_SPACEDIM==3)
         Uz[idx] * x[idx+zstride]+
         Lz[idx] * x[idx-zstride]+
#endif
         Ux[idx] * x[idx+1]+
         Lx[idx] * x[idx-1]+
         Uy[idx] * x[idx+ystride]+
         Ly[idx] * x[idx-ystride]);
 
}


extern "C" void kernel_gpu_apply(
 pointertype x, 
 pointertype xx_d,
 pointertype ax_d,
 pointertype alpha_d,
#if (BL_SPACEDIM==3)
 pointertype Uz_d,pointertype Lz_d,
#endif
 pointertype Ux_d, pointertype Uy_d, 
 pointertype Lx_d, pointertype Ly_d, 
 long int Np, pointertype Ax, int NX, int NY, int NZ) {

    long int N=Np;
    int zstride=NX*NY;
    int ystride=NX;
    long int grid_bytes=sizeof(sizetype)*N;

      /* grid dimensions divisible by 8 */
    int blocksIny = NY/4;
    dim3 dimBlock(512,1,1);
    dim3 dimGrid(NX/8,NZ*NY/64,1);

    hipMemcpy(xx_d, x, grid_bytes, hipMemcpyHostToDevice);

    gpu_apply <<< dimGrid, dimBlock >>> (xx_d,alpha_d, 
#if (BL_SPACEDIM==3)
      Uz_d,Lz_d,
#endif
      Ux_d, Uy_d, 
      Lx_d, Ly_d, 
      zstride,ystride, blocksIny, ax_d,
      N,NX,NY,NZ);

    hipMemcpy(Ax, ax_d, grid_bytes, hipMemcpyDeviceToHost);
}



// Function 1: Copy arrays to GPU and leave them there //////////////////////

extern "C" void kernel_wrapper(
 pointertype mask, 
 pointertype alpha, 
 pointertype alphasing, 
#if (BL_SPACEDIM==3)
 pointertype Uz,pointertype Lz,
#endif
 pointertype Ux, pointertype Uy, 
 pointertype Lx, pointertype Ly, 
 long int N, 
 pointertype* xx_d,
 pointertype* bb_d,
 pointertype* ax_d,
 pointertype* rhs_d,
 pointertype* soln_d,
 pointertype* red_d,
 pointertype* black_d,
 pointertype* mask_d,
 pointertype* alpha_d,
 pointertype* alphasing_d,
#if (BL_SPACEDIM==3)
 pointertype* Uz_d,pointertype* Lz_d,
#endif
 pointertype* Ux_d, pointertype* Uy_d,
 pointertype* Lx_d, pointertype* Ly_d) {

   long int grid_bytes=sizeof(sizetype)*N;

   // Allocate memory on GPU
   // if parameter passed as pointertype&, then (void **)&xx_d
   hipMalloc( (void **) xx_d, grid_bytes );
   hipMalloc( (void **) bb_d, grid_bytes );
   hipMalloc( (void **) ax_d, grid_bytes );

   hipMalloc( (void **) rhs_d, grid_bytes );
   hipMalloc( (void **) soln_d, grid_bytes );
   hipMalloc( (void **) red_d, grid_bytes );
   hipMalloc( (void **) black_d, grid_bytes );
   hipMalloc( (void **) mask_d, grid_bytes );

   hipMalloc( (void **) alpha_d, grid_bytes );
   hipMalloc( (void **) alphasing_d, grid_bytes );
#if (BL_SPACEDIM==3)
   cudaMalloc( (void **) Uz_d, grid_bytes );
   cudaMalloc( (void **) Lz_d, grid_bytes );
#endif
   hipMalloc( (void **) Ux_d, grid_bytes );
   hipMalloc( (void **) Uy_d, grid_bytes );
   hipMalloc( (void **) Lx_d, grid_bytes );
   hipMalloc( (void **) Ly_d, grid_bytes );

   // copy arrays from CPU to GPU
   hipMemcpy(*mask_d, mask, grid_bytes, hipMemcpyHostToDevice);
   hipMemcpy(*alpha_d, alpha, grid_bytes, hipMemcpyHostToDevice);
   hipMemcpy(*alphasing_d,alphasing,grid_bytes,hipMemcpyHostToDevice);
#if (BL_SPACEDIM==3)
   cudaMemcpy(*Uz_d, Uz, grid_bytes, cudaMemcpyHostToDevice);
   cudaMemcpy(*Lz_d, Lz, grid_bytes, cudaMemcpyHostToDevice);
#endif
   hipMemcpy(*Ux_d, Ux, grid_bytes, hipMemcpyHostToDevice);
   hipMemcpy(*Uy_d, Uy, grid_bytes, hipMemcpyHostToDevice);
   hipMemcpy(*Lx_d, Lx, grid_bytes, hipMemcpyHostToDevice);
   hipMemcpy(*Ly_d, Ly, grid_bytes, hipMemcpyHostToDevice);

  return;
}


// Function 3:  Free GPU /////////////////////////////////////////////////////

extern "C" void free_device(
pointertype* xx_d, 
pointertype* bb_d, 
pointertype* ax_d, 
pointertype* rhs_d,
pointertype* soln_d,
pointertype* red_d,
pointertype* black_d,
pointertype* mask_d,
pointertype* alpha_d, 
pointertype* alphasing_d, 
#if (BL_SPACEDIM==3)
pointertype* Uz_d,pointertype* Lz_d,
#endif
pointertype* Ux_d, pointertype* Uy_d, 
pointertype* Lx_d, pointertype* Ly_d)
{
 hipFree(*xx_d);
 hipFree(*bb_d);
 hipFree(*ax_d);
 hipFree(*rhs_d);
 hipFree(*soln_d);
 hipFree(*red_d);
 hipFree(*black_d);
 hipFree(*mask_d);
 hipFree(*alpha_d);
 hipFree(*alphasing_d);
#if (BL_SPACEDIM==3)
 cudaFree(*Uz_d);
 cudaFree(*Lz_d);
#endif
 hipFree(*Ux_d);
 hipFree(*Uy_d);
 hipFree(*Lx_d);
 hipFree(*Ly_d);
}

